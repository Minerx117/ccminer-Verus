#include "hip/hip_runtime.h"

#include <miner.h>
extern "C" {
#include <stdint.h>
#include <memory.h>
}
#define HARAKAS_RATE 32
#include <cuda_helper.h>
#define NPT 2
#define NBN 2
__device__  uint32_t sbox[64] =
{ 0x7b777c63, 0xc56f6bf2, 0x2b670130, 0x76abd7fe, 0x7dc982ca, 0xf04759fa, 0xafa2d4ad, 0xc072a49c, 0x2693fdb7, 0xccf73f36, 0xf1e5a534, 0x1531d871, 0xc323c704, 0x9a059618, 0xe2801207, 0x75b227eb, 0x1a2c8309, 0xa05a6e1b, 0xb3d63b52, 0x842fe329, 0xed00d153, 0x5bb1fc20, 0x39becb6a, 0xcf584c4a, 0xfbaaefd0, 0x85334d43, 0x7f02f945, 0xa89f3c50, 0x8f40a351, 0xf5389d92, 0x21dab6bc, 0xd2f3ff10, 0xec130ccd, 0x1744975f, 0x3d7ea7c4, 0x73195d64, 0xdc4f8160, 0x88902a22, 0x14b8ee46, 0xdb0b5ede, 0x0a3a32e0, 0x5c240649, 0x62acd3c2, 0x79e49591, 0x6d37c8e7, 0xa94ed58d, 0xeaf4566c, 0x08ae7a65, 0x2e2578ba, 0xc6b4a61c, 0x1f74dde8, 0x8a8bbd4b, 0x66b53e70, 0x0ef60348, 0xb9573561, 0x9e1dc186, 0x1198f8e1, 0x948ed969, 0xe9871e9b, 0xdf2855ce, 0x0d89a18c, 0x6842e6bf, 0x0f2d9941, 0x16bb54b0 };
__device__  uint32_t sbox[256] =
{0x63636363, 0x7c7c7c7c, 0x77777777, 0x7b7b7b7b, 0xf2f2f2f2, 0x6b6b6b6b, 0x6f6f6f6f, 0xc5c5c5c5, 0x30303030, 0x01010101, 0x67676767, 0x2b2b2b2b, 0xfefefefe, 0xd7d7d7d7, 0xabababab, 0x76767676, 0xcacacaca, 0x82828282, 0xc9c9c9c9, 0x7d7d7d7d, 0xfafafafa, 0x59595959, 0x47474747, 0xf0f0f0f0, 0xadadadad, 0xd4d4d4d4, 0xa2a2a2a2, 0xafafafaf, 0x9c9c9c9c, 0xa4a4a4a4, 0x72727272, 0xc0c0c0c0, 0xb7b7b7b7, 0xfdfdfdfd, 0x93939393, 0x26262626, 0x36363636, 0x3f3f3f3f, 0xf7f7f7f7, 0xcccccccc, 0x34343434, 0xa5a5a5a5, 0xe5e5e5e5, 0xf1f1f1f1, 0x71717171, 0xd8d8d8d8, 0x31313131, 0x15151515, 0x04040404, 0xc7c7c7c7, 0x23232323, 0xc3c3c3c3, 0x18181818, 0x96969696, 0x05050505, 0x9a9a9a9a, 0x07070707, 0x12121212, 0x80808080, 0xe2e2e2e2, 0xebebebeb, 0x27272727, 0xb2b2b2b2, 0x75757575, 0x09090909, 0x83838383, 0x2c2c2c2c, 0x1a1a1a1a, 0x1b1b1b1b, 0x6e6e6e6e, 0x5a5a5a5a, 0xa0a0a0a0, 0x52525252, 0x3b3b3b3b, 0xd6d6d6d6, 0xb3b3b3b3, 0x29292929, 0xe3e3e3e3, 0x2f2f2f2f, 0x84848484, 0x53535353, 0xd1d1d1d1, 0x00000000, 0xedededed, 0x20202020, 0xfcfcfcfc, 0xb1b1b1b1, 0x5b5b5b5b, 0x6a6a6a6a, 0xcbcbcbcb, 0xbebebebe, 0x39393939, 0x4a4a4a4a, 0x4c4c4c4c, 0x58585858, 0xcfcfcfcf, 0xd0d0d0d0, 0xefefefef, 0xaaaaaaaa, 0xfbfbfbfb, 0x43434343, 0x4d4d4d4d, 0x33333333, 0x85858585, 0x45454545, 0xf9f9f9f9, 0x02020202, 0x7f7f7f7f, 0x50505050, 0x3c3c3c3c, 0x9f9f9f9f, 0xa8a8a8a8, 0x51515151, 0xa3a3a3a3, 0x40404040, 0x8f8f8f8f, 0x92929292, 0x9d9d9d9d, 0x38383838, 0xf5f5f5f5, 0xbcbcbcbc, 0xb6b6b6b6, 0xdadadada, 0x21212121, 0x10101010, 0xffffffff, 0xf3f3f3f3, 0xd2d2d2d2, 0xcdcdcdcd, 0x0c0c0c0c, 0x13131313, 0xecececec, 0x5f5f5f5f, 0x97979797, 0x44444444, 0x17171717, 0xc4c4c4c4, 0xa7a7a7a7, 0x7e7e7e7e, 0x3d3d3d3d, 0x64646464, 0x5d5d5d5d, 0x19191919, 0x73737373, 0x60606060, 0x81818181, 0x4f4f4f4f, 0xdcdcdcdc, 0x22222222, 0x2a2a2a2a, 0x90909090, 0x88888888, 0x46464646, 0xeeeeeeee, 0xb8b8b8b8, 0x14141414, 0xdededede, 0x5e5e5e5e, 0x0b0b0b0b, 0xdbdbdbdb, 0xe0e0e0e0, 0x32323232, 0x3a3a3a3a, 0x0a0a0a0a, 0x49494949, 0x06060606, 0x24242424, 0x5c5c5c5c, 0xc2c2c2c2, 0xd3d3d3d3, 0xacacacac, 0x62626262, 0x91919191, 0x95959595, 0xe4e4e4e4, 0x79797979, 0xe7e7e7e7, 0xc8c8c8c8, 0x37373737, 0x6d6d6d6d, 0x8d8d8d8d, 0xd5d5d5d5, 0x4e4e4e4e, 0xa9a9a9a9, 0x6c6c6c6c, 0x56565656, 0xf4f4f4f4, 0xeaeaeaea, 0x65656565, 0x7a7a7a7a, 0xaeaeaeae, 0x08080808, 0xbabababa, 0x78787878, 0x25252525, 0x2e2e2e2e, 0x1c1c1c1c, 0xa6a6a6a6, 0xb4b4b4b4, 0xc6c6c6c6, 0xe8e8e8e8, 0xdddddddd, 0x74747474, 0x1f1f1f1f, 0x4b4b4b4b, 0xbdbdbdbd, 0x8b8b8b8b, 0x8a8a8a8a, 0x70707070, 0x3e3e3e3e, 0xb5b5b5b5, 0x66666666, 0x48484848, 0x03030303, 0xf6f6f6f6, 0x0e0e0e0e, 0x61616161, 0x35353535, 0x57575757, 0xb9b9b9b9, 0x86868686, 0xc1c1c1c1, 0x1d1d1d1d, 0x9e9e9e9e, 0xe1e1e1e1, 0xf8f8f8f8, 0x98989898, 0x11111111, 0x69696969, 0xd9d9d9d9, 0x8e8e8e8e, 0x94949494, 0x9b9b9b9b, 0x1e1e1e1e, 0x87878787, 0xe9e9e9e9, 0xcececece, 0x55555555, 0x28282828, 0xdfdfdfdf, 0x8c8c8c8c, 0xa1a1a1a1, 0x89898989, 0x0d0d0d0d, 0xbfbfbfbf, 0xe6e6e6e6, 0x42424242, 0x68686868, 0x41414141, 0x99999999, 0x2d2d2d2d, 0x0f0f0f0f, 0xb0b0b0b0, 0x54545454, 0xbbbbbbbb, 0x16161616}

__global__ void verus_gpu_hash(uint32_t threads, uint32_t startNonce, uint32_t *resNonce);
__device__ void haraka512_perm(unsigned char *out, unsigned char *in);
static uint32_t *d_nonces[MAX_GPUS];
__constant__ uint8_t blockhash_half[128];
__constant__ uint32_t ptarget[8];

__device__   void memcpy_decker(unsigned char *dst, unsigned char *src, int len) {
	int i;
	for (i = 0; i< len; i++) { dst[i] = src[i]; }
}

__host__
void verus_init(int thr_id)
{
CUDA_SAFE_CALL(hipMalloc(&d_nonces[thr_id], 2 * sizeof(uint32_t)));
};
void verus_setBlock(void *blockf, const void *pTargetIn)
{
CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ptarget), pTargetIn, 8 * sizeof(uint32_t), 0, hipMemcpyHostToDevice));
CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(blockhash_half), blockf, 64 * sizeof(uint8_t), 0, hipMemcpyHostToDevice));
};

__host__
void verus_hash(int thr_id, uint32_t threads, uint32_t startNonce, uint32_t *resNonces)
{
	hipMemset(d_nonces[thr_id], 0xff, 2 * sizeof(uint32_t));
	const uint32_t threadsperblock = 256;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	verus_gpu_hash << <grid, block >> >(threads, startNonce, d_nonces[thr_id]);
	//hipDeviceSynchronize();
	hipMemcpy(resNonces, d_nonces[thr_id], NBN * sizeof(uint32_t), hipMemcpyDeviceToHost);
	//memcpy(resNonces, h_nonces[thr_id], NBN * sizeof(uint32_t));

};



//__constant__ static const
#define XT(x) (((x) << 1) ^ ((((x) >> 7) & 1) * 0x1b))

// Simulate _mm_aesenc_si128 instructions from AESNI
__device__   void aesenc(uint32_t *s,uint32_t *sharedMemory1)
{
	uint32_t i, t, u;
	register uint32_t  v[4][4];
	
	for (i = 0; i < 16; ++i) {
		v[((i >> 2) + 4 - (i & 3)) & 3][i & 3] = sharedMemory1[s[i]];
	
	}

	for (i = 0; i < 4; ++i) {
		t = v[i][0];
		u = v[i][0] ^ v[i][1] ^ v[i][2] ^ v[i][3];
		v[i][0] = v[i][0] ^ u ^ XT(v[i][0] ^ v[i][1]);


		v[i][0] = v[i][0] ^ u ^ XT(v[i][0] ^ v[i][1]);

		v[i][1] = v[i][1] ^ u ^ XT(v[i][1] ^ v[i][2]);
		v[i][2] = v[i][2] ^ u ^ XT(v[i][2] ^ v[i][3]);
		v[i][3] = v[i][3] ^ u ^ XT(v[i][3] ^ t);
	}
	for (i = 0; i < 16; ++i) {
		s[i] = v[i >> 2][i & 3]; // VerusHash have 0 rc vector
	}
}

// Simulate _mm_unpacklo_epi32
__device__ __forceinline__   void unpacklo32(unsigned char *t, unsigned char *a, unsigned char *b)
{
	unsigned char tmp[16];
	memcpy_decker(tmp, a, 4);
	memcpy_decker(tmp + 4, b, 4);
	memcpy_decker(tmp + 8, a + 4, 4);
	memcpy_decker(tmp + 12, b + 4, 4);
	memcpy_decker(t, tmp, 16);
}

// Simulate _mm_unpackhi_epi32
__device__  __forceinline__  void unpackhi32(unsigned char *t, unsigned char *a, unsigned char *b)
{
	unsigned char tmp[16];
	memcpy_decker(tmp, a + 8, 4);
	memcpy_decker(tmp + 4, b + 8, 4);
	memcpy_decker(tmp + 8, a + 12, 4);
	memcpy_decker(tmp + 12, b + 12, 4);
	memcpy_decker(t, tmp, 16);

}


__global__ __launch_bounds__(256, 2)
void verus_gpu_hash(uint32_t threads, uint32_t startNonce, uint32_t *resNonce)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x) * 4;
	
	int i, j; 
	uint32_t s[64] = { 0 };
	uint32_t tmp[16];
		__shared__ uint32_t sharedMemory1[64];
	if (threadIdx.x < 64)
		sharedMemory1[threadIdx.x] = sbox[threadIdx.x];//	for (i = 0; i < 64; ++i)
					

	uint32_t nounce[4];
	uint64_t in[4];
	nounce[0] = startNonce + thread;
	nounce[1] = startNonce + thread + 1;
	nounce[2] = startNonce + thread + 2;
	nounce[3] = startNonce + thread + 3;
		
		
		uint64_t blockhash[4];
		for (i = 0; i < 32; ++i) {
			((uint8_t*)&s[i])[0] = blockhash_half[i];
			((uint8_t*)&s[i])[1] = blockhash_half[i];
			((uint8_t*)&s[i])[2] = blockhash_half[i];
			((uint8_t*)&s[i])[3] = blockhash_half[i];
		}
		for (i = 32; i < 64; ++i) {
			((uint8_t*)&s[i])[0] = 0;
			((uint8_t*)&s[i])[1] = 0;
			((uint8_t*)&s[i])[2] = 0;
			((uint8_t*)&s[i])[3] = 0;
		}
		for (i = 32; i < 40; ++i) {
			((uint8_t*)&s[i])[0] = ((uint8_t*)&nounce)[i];
			((uint8_t*)&s[i])[1] = ((uint8_t*)&nounce)[i];
			((uint8_t*)&s[i])[2] = ((uint8_t*)&nounce)[i];
			((uint8_t*)&s[i])[3] = ((uint8_t*)&nounce)[i];
		}
		//memcpy(s, blockhash_half, 32);
		//memset(s + 32, 0x0, 32);
		//((uint32_t *)&s)[8] = startNonce + thread;
		//memcpy(in +48, s + 48, 8);
		//memcpy_decker(s, in, 64);


    #pragma unroll 
		for (i = 0; i < 5; ++i) {
			// aes round(s)
			//__syncthreads();
			for (j = 0; j < 2; ++j) {

				aesenc(s, sharedMemory1);
				aesenc(s + 16, sharedMemory1);
				aesenc(s + 32, sharedMemory1);
				aesenc(s + 48, sharedMemory1);
			}
			unpacklo32(tmp, s, s + 16);
			unpackhi32(s, s, s + 16);
			unpacklo32(s + 16, s + 32, s + 48);
			unpackhi32(s + 32, s + 32, s + 48);
			unpacklo32(s + 48, s, s + 32);
			unpackhi32(s, s, s + 32);
			unpackhi32(s + 32, s + 16, tmp);
			unpacklo32(s + 16, s + 16, tmp);

		}
		for (i = 48; i < 56; i++) {
			s[i] = s[i] ^ in[i];
		}

		memcpy_decker((unsigned char*)blockhash + 24, s + 48, 8);
		
		

		if (blockhash[3] < ((uint64_t*)&ptarget)[3]) { resNonce[0] = nounce; }
	
};


